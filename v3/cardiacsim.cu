#include "hip/hip_runtime.h"
/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 */
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "cuPrintf.cu"
using namespace std;


// Utilities
// 

// Timer
// Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime()
{
  struct timeval TV;
  struct timezone TZ;

  const int RC = gettimeofday(&TV, &TZ);
  if(RC == -1) {
    cerr << "ERROR: Bad call to gettimeofday" << endl;
    return(-1);
  }

  return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}  // end getTime()

// Allocate a 2D array
double **alloc2D(int m,int n){
  double **E;
  int nx=n, ny=m;
  E = (double**)malloc(sizeof(double*)*ny + sizeof(double)*nx*ny);
  assert(E);
  int j;
  for(j=0;j<ny;j++) 
    E[j] = (double*)(E+ny) + j*nx;
  return(E);
}

// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
double stats(double **E, int m, int n, double *_mx){
  double mx = -1;
  double l2norm = 0;
  int i, j;
  for (j=1; j<=m; j++)
    for (i=1; i<=n; i++) {
      l2norm += E[j][i]*E[j][i];
      if (E[j][i] > mx)
        mx = E[j][i];
    }
  *_mx = mx;
  l2norm /= (double) ((m)*(n));
  l2norm = sqrt(l2norm);
  return l2norm;
}

// External functions
extern "C" {
  void splot(double **E, double T, int niter, int m, int n);
}
void cmdLine(int argc, char *argv[], double& T, int& n, int& px, int& py, int& plot_freq, int& no_comm, int&num_threads, int& block_size);

__global__ 
void KernelMethod(double* E, double *E_prev, double* R, const int N, 
                  double alpha, double dt, double kk, double a, double b, 
                  double epsilon, double M1, double M2) {

  int row = blockIdx.y*blockDim.y + threadIdx.y+1;
  int col = blockIdx.x*blockDim.x + threadIdx.x+1;
  
  if ((row < N+1) && (col < N+1)) {
    // Left ghost region
    E_prev[row*(N+2)]  = E_prev[row*(N+2) + 2];
    // Right ghost region
    E_prev[row*(N+2)+(N+1)] = E_prev[row*(N+2)+(N-1)];
    // Top ghost region
    E_prev[col] = E_prev[2*(N+2)+col];
    // Bottom ghost region
    E_prev[(N+1)*(N+2)+col] = E_prev[(N-1)*(N+2)+col];
  
    int j = row*(N+2), i = col;

    E[row*(N+2)+col] = E_prev[row*(N+2)+col]+alpha*(E_prev[row*(N+2)+col+1]+E_prev[row*(N+2)+col-1]-4*E_prev[row*(N+2)+col]+E_prev[(row-1)*(N+2)+col]+E_prev[(row+1)*(N+2)+col]);
    
    double e = E[j+i]; 
    double r = R[j+i];

    e = e -dt*(kk* e *(e-a)*(e-1)+ e *r);
    r = r + dt*(epsilon+M1* r/( e+M2))*(-r-kk* e*(e-b-1));
     
    E[i+j] = e;
    R[i+j] = r;
   
    }

}


void simulate (const double alpha, const int n, const int m, const double kk,
    const double dt, const double a, const double epsilon,
    const double M1,const double  M2, const double b, 
    double* E_cuda, double* E_prev_cuda, double* R_cuda, const int block_size)
{
  /*int i, j; */
  /* 
   * Copy data from boundary of the computational box 
   * to the padding region, set up for differencing
   * on the boundary of the computational box
   * Using mirror boundaries
   */

  dim3 tblock(block_size, block_size, 1);
  dim3 grid(ceil(m*1.0/tblock.x), ceil(m*1.0/tblock.y));

  KernelMethod<<<grid, tblock>>>(E_cuda, E_prev_cuda, R_cuda, m, alpha, dt, kk, a, b, epsilon, M1, M2);
  hipDeviceSynchronize();
}

void unFlattenArray(double * flat, double ** ary, int N) {
  int j, i;
  for (j=1; j<N+2; j++) {
    for (i=1; i<N+2; i++) {
      ary[j][i] = flat[j*(N+2)+i];
    }
  }
}

// Main program
int main (int argc, char** argv)
{
  /*
   *  Solution arrays
   *   E is the "Excitation" variable, a voltage
   *   R is the "Recovery" variable
   *   E_prev is the Excitation variable for the previous timestep,
   *      and is used in time integration
   */
  double **E, **R, **E_prev;

  // Various constants - these definitions shouldn't change
  const double a=0.1, b=0.1, kk=8.0, M1= 0.07, M2=0.3, epsilon=0.01, d=5e-5;

  double T=1000.0;
  int m=200,n=200;
  int plot_freq = 0;
  int px = 1, py = 1;
  int no_comm = 0;
  int num_threads=1; 
  int block_size = 16;

  cmdLine( argc, argv, T, n,px, py, plot_freq, no_comm, num_threads, block_size);
  m = n;  
  // Allocate contiguous memory for solution arrays
  // The computational box is defined on [1:m+1,1:n+1]
  // We pad the arrays in order to facilitate differencing on the 
  // boundaries of the computation box
  E = alloc2D(m+2,n+2);
  E_prev = alloc2D(m+2,n+2);
  R = alloc2D(m+2,n+2);

  printf("Block size: %d\n", block_size);


  int i,j;
  // Initialization
  for (j=1; j<=m; j++)
    for (i=1; i<=n; i++) 
      E_prev[j][i] = R[j][i] = 0;

  for (j=1; j<=m; j++)
    for (i=n/2+1; i<=n; i++)
      E_prev[j][i] = 1.0;

  for (j=m/2+1; j<=m; j++) 
    for (i=1; i<=n; i++) 
      R[j][i] = 1.0;

  double * E_cuda; 
  double * R_cuda;
  double * E_prev_cuda;

  double * E_flat = (double *) malloc(sizeof(double) * (m+2) * (n+2));
  double * R_flat = (double *) malloc(sizeof(double) * (m+2) * (n+2));
  double * E_prev_flat = (double *) malloc(sizeof(double) * (m+2) * (n+2));

  for (j=1; j<=m; j++) {
    for (i=1; i<=n;i++) {
      E_prev_flat[(j*(n+2))+i] = E_prev[j][i];
      R_flat[(j*(n+2))+i] = R[j][i];
    }
  }

  hipMalloc((void**) &E_cuda, (m+2)*(m+2)*sizeof(double));
  hipMalloc((void**) &R_cuda, (m+2)*(m+2)*sizeof(double));
  hipMalloc((void**) &E_prev_cuda, (m+2)*(m+2)*sizeof(double));

  hipMemcpy(E_cuda, E_flat, (m+2)*(n+2)*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(R_cuda, R_flat, (m+2)*(n+2)*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(E_prev_cuda, E_prev_flat, (m+2)*(n+2)*sizeof(double), hipMemcpyHostToDevice);

  double dx = 1.0/n;
  // For time integration, these values shouldn't change 
  double rp= kk*(b+1)*(b+1)/4;
  double dte=(dx*dx)/(d*4+((dx*dx))*(rp+kk));
  double dtr=1/(epsilon+((M1/M2)*rp));
  double dt = (dte<dtr) ? 0.95*dte : 0.95*dtr;
  double alpha = d*dt/(dx*dx);

  cout << "Grid Size       : " << n << endl; 
  cout << "Duration of Sim : " << T << endl; 
  cout << "Time step dt    : " << dt << endl; 
  cout << "Process geometry: " << px << " x " << py << endl;
  if (no_comm)
    cout << "Communication   : DISABLED" << endl;

  cout << endl;

  // Start the timer
  double t0 = getTime();

  // Simulated time is different from the integer timestep number
  // Simulated time
  double t = 0.0;
  // Integer timestep number
  int niter=0;


  while (t<T) {

    t += dt;
    niter++;

    simulate(alpha, n, m, kk, dt, a, epsilon, M1, M2, b, E_cuda, E_prev_cuda, R_cuda, block_size); 

    //swap current E with previous E
    double *tmp = E_cuda; E_cuda = E_prev_cuda; E_prev_cuda = tmp;

    if (plot_freq){
      int k = (int)(t/plot_freq);
      if ((t - k * plot_freq) < dt){
        splot(E,t,niter,m+2,n+2);
      }
    }
  }//end of while loop

  hipMemcpy(E_flat, E_cuda, (m+2)*(n+2)*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(R_flat, R_cuda, (m+2)*(n+2)*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(E_prev_flat, E_prev_cuda, (m+2)*(n+2)*sizeof(double), hipMemcpyDeviceToHost);

  double time_elapsed = getTime() - t0;

  unFlattenArray(E_flat, E, m);
  unFlattenArray(E_prev_flat, E_prev, m);
  unFlattenArray(R_flat, R, m);

  double Gflops = (double)(niter * (1E-9 * n * n ) * 28.0) / time_elapsed ;
  double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0  ))/time_elapsed;

  cout << "Number of Iterations        : " << niter << endl;
  cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
  cout << "Sustained Gflops Rate       : " << Gflops << endl; 
  cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl; 

  double mx;
  double l2norm = stats(E_prev,m,n,&mx);
  cout << "Max: " << mx <<  " L2norm: "<< l2norm << endl;

  if (plot_freq){
    cout << "\n\nEnter any input to close the program and the plot..." << endl;
    getchar();
  }

  free (E);
  free (E_prev);
  free (R);

  free (E_flat);
  free (E_prev_flat);
  free (R_flat);

  hipFree(E_cuda);
  hipFree(E_prev_cuda);
  hipFree(R_cuda);
  return 0;
}
